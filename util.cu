#include "eq.h"

void safely_call(hipError_t err, const char *msg, const char *file_name,
                 const int line_number) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg,
            file_name, line_number, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}